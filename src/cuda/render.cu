#include "hip/hip_runtime.h"
#include "render.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "physics.h"
#include "raytrace.cuh"
#include "vector-operators.h"
#include "cuda-tools.cuh"
#include "cuda-vbo.cuh"
#include "mathParser.hpp"
#include "draw.cuh"

#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

#define SIG_HIGHLIGHT_COLOR float4{0.5, 1.0, 0.5, 0.1}
#define MAT_HIGHLIGHT_COLOR float4{1.0, 0.5, 0.5, 0.1}

//// RENDERING ////

template<typename T>
__global__ void renderFieldEM_k(FluidField<T> src, CudaTexture dst, RenderParams<T> rp, FluidParams<T> cp)
{
  typedef typename DimType<T,3>::VEC_T VT3;
  typedef typename DimType<T,4>::VEC_T VT4;
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  if(ix < dst.size.x && iy < dst.size.y)
    {
      int ti = ix + iy*dst.size.x;
      int2 fp = int2{(int)(ix*(src.size.x/(T)dst.size.x)),  // scale texture index to field index
                     (int)(iy*(src.size.y/(T)dst.size.y))};
      
      VT4 color = VT4{0.0, 0.0, 0.0, 0.0};
      for(int iz = max(0, min(src.size.z-1, rp.zRange.y)); iz >= rp.zRange.x; iz--)
        {
          int fi = src.idx(fp.x, fp.y, iz);
          T qLen = (src.Qp[fi] - src.Qn[fi]); T eLen = length(src.E[fi]); T bLen = length(src.B[fi]);
          VT4 col = rp.emBrightness*rp.emOpacity*(qLen*rp.getFinalColor(FLUID_RENDER_Q) +
                                                  eLen*rp.getFinalColor(FLUID_RENDER_E) +
                                                  bLen*rp.getFinalColor(FLUID_RENDER_B));
          // VT4 col = renderCell(src, fi, rp);

          VT3 pCell = VT3{(T)fp.x, (T)fp.y, (T)iz}; VT3 pSrc = rp.penPos;
          VT3 diff; VT3 diff0; VT3 diff1;  VT3 dist_2; VT3 dist0_2; VT3 dist1_2;
          if(rp.sigPenHighlight &&
             penOverlap3 (pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.sigPen, cp, 0.0f) &&
             !penOverlap3(pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.sigPen, cp, -1.0f)) { col += SIG_HIGHLIGHT_COLOR; }
          if(rp.matPenHighlight &&
             penOverlap3 (pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.matPen, cp, 0.0f) &&
             !penOverlap3(pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.matPen, cp, -1.0f)) { col += MAT_HIGHLIGHT_COLOR; }
          
          fluidBlend(color, col, rp);
          if(color.x >= 1 || color.y >= 1 || color.z >= 1) { break; }
        }
      // blend with background color
      T a = color.w;
      color += VT4{BG_COLOR.x, BG_COLOR.y, BG_COLOR.z, 0.0} * BG_COLOR.w*(1-a*rp.emBrightness);
      color.w += BG_COLOR.w*(1-color.w)*(rp.emOpacity);
      dst[ti] += float4{ max(0.0f, min(1.0f, (float)color.x)), max(0.0f, min(1.0f, (float)color.y)), max(0.0f, min(1.0f, (float)color.z)), 1.0f };
    }
}

template<typename T>
__global__ void renderFieldMat_k(Field<Material<T>> src, CudaTexture dst, RenderParams<T> rp, FluidParams<T> cp)
{
  typedef typename DimType<T,3>::VEC_T VT3;
  typedef typename DimType<T,4>::VEC_T VT4;
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  if(ix < dst.size.x && iy < dst.size.y)
    {
      int ti = ix + iy*dst.size.x;
      int2 fp = int2{(int)(ix*(src.size.x/(T)dst.size.x)),  // scale texture index to field index
                     (int)(iy*(src.size.y/(T)dst.size.y))};
      
      VT4 color = VT4{0.0f, 0.0f, 0.0f, 0.0f};
      for(int iz = min(src.size.z-1, rp.zRange.y); iz >= rp.zRange.x; iz--)
        {
          int fi = src.idx(fp.x, fp.y, iz);
          Material<T> mat = src[fi];
          VT4 col = (mat.vacuum() ? VT4{0.0f, 0.0f, 0.0f, 1.0f} :
                     rp.matBrightness*rp.matOpacity*(mat.permittivity*rp.getFinalColor(FLUID_RENDER_EP) +
                                                     mat.permeability*rp.getFinalColor(FLUID_RENDER_MU) +
                                                     mat.conductivity*rp.getFinalColor(FLUID_RENDER_SIG)));
         
          VT3 pCell = VT3{(T)fp.x, (T)fp.y, (T)iz}; VT3 pSrc = rp.penPos;
          VT3 diff; VT3 diff0; VT3 diff1;  VT3 dist_2; VT3 dist0_2; VT3 dist1_2;
          if(rp.sigPenHighlight &&
             penOverlap3 (pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.sigPen, cp, 0.0f) &&
             !penOverlap3(pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.sigPen, cp, -1.0f)) { col += SIG_HIGHLIGHT_COLOR; }
          if(rp.matPenHighlight &&
             penOverlap3 (pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.matPen, cp, 0.0f) &&
             !penOverlap3(pCell, pSrc, diff, diff0, diff1, dist_2, dist0_2, dist1_2, (Pen<T>*)&rp.matPen, cp, -1.0f)) { col += MAT_HIGHLIGHT_COLOR; }

          fluidBlend(color, col, rp);
          if(color.x >= 1.0f || color.y >= 1.0f || color.z >= 1.0f) { break; }
        }
      // blend with background color
      T a = color.w;
      color += VT4{BG_COLOR.x, BG_COLOR.y, BG_COLOR.z, 0.0} * BG_COLOR.w*(1-a*rp.matBrightness);
      color.w += BG_COLOR.w*(1-color.w)*(rp.matOpacity);
      dst[ti] += float4{ max(0.0f, min(1.0f, (float)color.x)), max(0.0f, min(1.0f, (float)color.y)), max(0.0f, min(1.0f, (float)color.z)), 1.0f };
    }
}


template<typename T>
__global__ void rtRenderFieldEM_k(FluidField<T> src, CudaTexture dst, CameraDesc<T> cam, RenderParams<T> rp, FluidParams<T> cp,
                                  typename DimType<T, 2>::VEC_T aspect)
{
  typedef typename DimType<T,2>::VEC_T VT2;
  typedef typename DimType<T,4>::VEC_T VT4;
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  if(ix < dst.size.x && iy < dst.size.y)
    {
      Ray<T> ray = cam.castRay(VT2{ix/(T)dst.size.x, iy/(T)dst.size.y}, aspect);
      ray.pos /= cp.u.dL; // scale by cell size
      VT4 color = rayTraceField(src, ray, rp, cp);
      
      long ti = ix + iy*dst.size.x;
      dst[ti] += float4{ max(0.0f, min(1.0f, (float)color.x)), max(0.0f, min(1.0f, (float)color.y)), max(0.0f, min(1.0f, (float)color.z)), 1.0f };
    }
}
template<typename T>
__global__ void rtRenderFieldMat_k(FluidField<T> src, CudaTexture dst, CameraDesc<T> cam, RenderParams<T> rp, FluidParams<T> cp,
                                   typename DimType<T, 2>::VEC_T aspect)
{
  typedef typename DimType<T,2>::VEC_T VT2;
  typedef typename DimType<T,4>::VEC_T VT4;
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  if(ix < dst.size.x && iy < dst.size.y)
    {
      Ray<T> ray = cam.castRay(VT2{ix/(T)dst.size.x, iy/(T)dst.size.y}, aspect);
      ray.pos /= cp.u.dL; // scale by cell size
      VT4 color = rayTraceField(src, ray, rp, cp);
      
      long ti = ix + iy*dst.size.x;
      dst[ti] += float4{ max(0.0f, min(1.0f, (float)color.x)), max(0.0f, min(1.0f, (float)color.y)), max(0.0f, min(1.0f, (float)color.z)), 1.0f };
    }
}

// wrappers
template<typename T>
void renderFieldEM(FluidField<T> &src, CudaTexture &dst, const RenderParams<T> &rp, const FluidParams<T> &cp)
{
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y)); // 2D -- thread texture pixels
      bool mapped = dst.mapped;
      if(!mapped) { dst.map(); }
      renderFieldEM_k<<<grid, threads>>>(src, dst, rp, cp);
      if(!mapped) { dst.unmap(); }
    }
  else { std::cout << "Skipped EMField render --> " << src.size << " / " << dst.size << " \n"; }
}
template<typename T>
void renderFieldMat(Field<Material<T>> &src, CudaTexture &dst, const RenderParams<T> &rp, const FluidParams<T> &cp)
{
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y)); // 2D -- thread texture pixels
      bool mapped = dst.mapped;
      if(!mapped) { dst.map(); }
      renderFieldMat_k<<<grid, threads>>>(src, dst, rp, cp);
      if(!mapped) { dst.unmap(); }
    }
  else { std::cout << "Skipped EMField render --> " << src.size << " / " << dst.size << " \n"; }
}

template<typename T>
void raytraceFieldEM(FluidField<T> &src, CudaTexture &dst, const Camera<T> &camera, const RenderParams<T> &rp, const FluidParams<T> &cp, 
                     const Vector<T, 2> &aspect)
{
  typedef typename DimType<T,2>::VEC_T VT2;
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y)); // 2D -- thread texture pixels
      bool mapped = dst.mapped;
      if(!mapped) { dst.map(); }
      rtRenderFieldEM_k<<<grid, threads>>>(src, dst, camera.desc, rp, cp, VT2{aspect.x, aspect.y});
      if(!mapped) { dst.unmap(); }
    }
  else { std::cout << "Skipped EMField render (RT) --> " << src.size << " / " << dst.size << " \n"; }
}

template<typename T>
void raytraceFieldMat(FluidField<T> &src, CudaTexture &dst, const Camera<T> &camera, const RenderParams<T> &rp, const FluidParams<T> &cp,
                      const Vector<T, 2> &aspect)
{
  typedef typename DimType<T,2>::VEC_T VT2;
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y)); // 2D -- thread texture pixels
      bool mapped = dst.mapped;
      if(!mapped) { dst.map(); }
      rtRenderFieldMat_k<<<grid, threads>>>(src, dst, camera.desc, rp, cp, VT2{aspect.x, aspect.y});
      if(!mapped) { dst.unmap(); }
    }
  else { std::cout << "Skipped EMField render (RT) --> " << src.size << " / " << dst.size << " \n"; }
}

// template instantiation
template void renderFieldEM   <float>(FluidField<float>      &src, CudaTexture &dst, const RenderParams<float> &rp, const FluidParams<float> &cp);
template void renderFieldMat  <float>(Field<Material<float>> &src, CudaTexture &dst, const RenderParams<float> &rp, const FluidParams<float> &cp);
template void raytraceFieldEM <float>(FluidField<float>      &src, CudaTexture &dst, const Camera<float> &camera,
                                      const RenderParams<float> &rp, const FluidParams<float> &cp, const Vec2f &aspect);
template void raytraceFieldMat<float>(FluidField<float>      &src, CudaTexture &dst, const Camera<float> &camera,
                                      const RenderParams<float> &rp, const FluidParams<float> &cp, const Vec2f &aspect);
