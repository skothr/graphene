#include "hip/hip_runtime.h"
#include "field.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "physics.h"
#include "raytrace.cuh"
#include "vector-operators.h"
#include "cuda-tools.cuh"
#include "mathParser.hpp"

#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 8


//// FILL FIELD ////

// fills field with constant value
template<typename T> __global__ void fillFieldValue_k(Field<T> dst, T val)
{
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix >= 0 && iy >= 0 && ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long i = dst.idx(ix, iy, iz);
      dst[i] = val;
    }
}

// fills field with constant value
template<typename T>
__global__ void fillFieldMaterial_k(Field<Material<T>> dst, CudaExpression<T> *dExprEp, CudaExpression<T> *dExprMu, CudaExpression<T> *dExprSig)
{
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix >= 0 && iy >= 0 && ix < dst.size.x && iy < dst.size.y && iz < dst.size.z && dExprEp && dExprMu && dExprSig)
    {
      unsigned long i = dst.idx(ix, iy, iz);
      Material<T> M = dst[i];
      
      // allowed expression variables
      float3 s = float3{(float)dst.size.x, (float)dst.size.y, (float)dst.size.z}; // (s --> size)
      float3 p = float3{(float)ix, (float)iy, (float)iz};                         // (p --> position)
      float3 c = p - s/2.0;              // offset from field center
      float3 n = normalize(c);           // unit vector from field center
      float  r = length(c);              // (r --> radius) distance from field center
      float  t = (float)atan2(n.y, n.x); // (t --> theta)  angle measured from field center

      // run expression
      const int nVars = 8;
      T vars[nVars]; // {"px", "py", "pz", "sx", "sy", "sz", "r", "t"}
      vars[0] = p.x; vars[1] = p.y; vars[2] = p.z; // "px" / "py" / "pz"
      vars[3] = s.x; vars[4] = s.y; vars[5] = s.z; // "sx" / "sy" / "sz"
      vars[6] = r;   vars[7] = t;                  // "r" / "t"
      
      M.ep   = dExprEp->calculate(vars);
      M.mu   = dExprMu->calculate(vars);
      M.sig  = dExprSig->calculate(vars);
      M.nonVacuum = true;
      dst[i] = M;
    }
}

// fills field via given math expression
template<typename T> __global__ void fillField_k(Field<T> dst, CudaExpression<T> *expr);
template<typename T> __global__ void fillFieldChannel_k(Field<T> dst, CudaExpression<typename Dim<T>::BASE_T> *expr, int channel=-1);
template<> __global__ void fillField_k<float>(Field<float> dst, CudaExpression<float> *expr)
{
  unsigned long ix = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned long iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long i = dst.idx(ix, iy, iz);

      // allowed expression variables
      float3 s = float3{(float)dst.size.x, (float)dst.size.y, (float)dst.size.z}; // (s --> size)
      float3 p = float3{(float)ix, (float)iy, (float)iz};                         // (p --> position)
      float3 c = p - s/2.0;              // offset from field center
      float3 n = normalize(c);           // unit vector from field center
      float  r = length(c);              // (r --> radius) distance from field center
      float  t = (float)atan2(n.y, n.x); // (t --> theta)  angle measured from field center

      // run expression
      const int nVars = 8;
      float vars[nVars]; // {"px", "py", "pz", "sx", "sy", "sz", "r", "t"}
      vars[0] = p.x; vars[1] = p.y; vars[2] = p.z; // "px" / "py" / "pz"
      vars[3] = s.x; vars[4] = s.y; vars[5] = s.z; // "sx" / "sy" / "sz"
      vars[6] = r;   vars[7] = t;                  // "r" / "t"
      dst[i] = expr->calculate(vars);
    }
}

template<> __global__ void fillField_k<float3>(Field<float3> dst, CudaExpression<float3> *expr)
{
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long i = dst.idx(ix, iy,iz);

      // allowed expression variables
      float3 s    = makeV<float3>(dst.size); //float3{(float)dst.size.x, (float)dst.size.y, 1.0}; // (s --> size)
      float3 p    = float3{(float)ix, (float)iy, (float)iz};  // (p --> position)
      float3 c    = p - s/2.0;       // offset from field center
      float3 n    = normalize(c);    // unit vector from field center
      float  r    = length(c);       // (r --> radius) distance from field center
      float  t    = atan2(n.y, n.x); // (t --> theta)  angle measured from field center (cylindrical)
      
      const int nVars = 5;
      float3 vars[nVars]; // {"p", "s", "r", "n", "t"}
      vars[0] = p;               // "p" -- position from origin (field index 0)
      vars[1] = s;               // "s" -- size
      vars[2] = c;               // "r" -- radius (position from center)
      vars[3] = n;               // "n" -- normalized radius
      //vars[4] = float3{t, t, t}; // "t" -- theta from center
      vars[4] = float3{atan2(n.z, n.y),
                       atan2(n.x, n.z),
                       atan2(n.y, n.x)}; // t alternative?
      // calculate value
      dst[i] = expr->calculate(vars);
    }
}
// only set one component/channel of each cell (used for setting +/- charge in Q.x/y
template<> __global__ void fillFieldChannel_k<float2>(Field<float2> dst, CudaExpression<float> *expr, int channel)
{
  unsigned long ix = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned long iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long i = dst.idx(ix, iy, iz);
      // allowed expression variables
      float3 s    = float3{(float)dst.size.x, (float)dst.size.y, (float)dst.size.z}; // (s --> size)
      float3 p    = float3{(float)ix, (float)iy, (float)iz};           // (p --> position)
      float3 c    = p - s/2.0;                                         // offset from field center
      float3 n    = normalize(c);                                      // unit vector from field center
      float  r    = length(c);                                         // (r --> radius) distance from field center
      float  t    = atan2(n.y, n.x);                                   // (t --> theta)  angle measured from field center
      
      // run expression
      const int nVars = 8;
      float vars[nVars]; // {"px", "py", "pz", "sx", "sy", "sz", "r", "t"}
      vars[0] = p.x; vars[1] = p.y; vars[2] = p.z; // "px" / "py" / "pz"
      vars[3] = s.x; vars[4] = s.y; vars[5] = s.z; // "sx" / "sy" / "sz"
      vars[6] = r;   vars[7] = t;                  // "r" / "t"
      float val = expr->calculate(vars);

      // write to channel
      if     (channel == 0) { dst[i].x = val; }
      else if(channel == 1) { dst[i].y = val; }
      else                  { dst[i] = float2{val, val}; }
    }
}

template<typename T>
void fillFieldValue(Field<T> &dst, const T &val)
{
  if(dst.size.x > 0 && dst.size.y > 0 && dst.size.z > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      fillFieldValue_k<<<grid, threads>>>(dst, val);
    }
  else { std::cout << "Skipped Field<float> fill --> " << dst.size << " --> " << val << "\n"; }
}

// wrapppers
template<typename T>
void fillField(Field<T> &dst, CudaExpression<T> *dExpr)
{
  if(dst.size.x > 0 && dst.size.y > 0 && dst.size.z > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      if(dExpr) { fillField_k<<<grid, threads>>>(dst, dExpr); }
      else { std::cout << "====> WARNING: fillField skipped -- null expression pointer ("
                       << "dExpr: "<< (long)((void*)dExpr) << ")\n"; }
    }
  else { std::cout << "Skipped Field<float> fill --> " << dst.size << " --> " << (long)((void*)dExpr) << "\n"; }
}

template<typename T>
void fillFieldMaterial<T>(Field<Material<T>> &dst, CudaExpression<T> *dExprEp, CudaExpression<T> *dExprMu, CudaExpression<T> *dExprSig)
{
  if(dst.size.x > 0 && dst.size.y > 0 && dst.size.z > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      if(dExprEp && dExprMu && dExprSig) { fillFieldMaterial_k<<<grid, threads>>>(dst, dExprEp, dExprMu, dExprSig); }
      else
        {
          std::cout << "====> WARNING: fillFieldMateral skipped -- null expression pointer ("
                    << "ε: "<< (long)((void*)dExprEp) << "|μ: " << (long)((void*)dExprMu) << "|σ: " << (long)((void*)dExprSig) << ")\n";
        }
    }
  else { std::cout << "Skipped Field<float> Material fill --> " << dst.size << " \n"; }
}
template<typename T>
void fillFieldChannel(Field<T> &dst, CudaExpression<typename Dim<T>::BASE_T> *dExpr, int channel)
{
  if(dst.size.x > 0 && dst.size.y > 0 && dst.size.z > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      if(dExpr) { fillFieldChannel_k<<<grid, threads>>>(dst, dExpr, channel); }
    }
  else { std::cout << "Skipped Field<float> fill --> " << dst.size << " \n"; }
}

// template instantiation
template void fillFieldMaterial<float >         (Field<Material<float>> &dst,
                                                 CudaExpression<float> *dExprEp, CudaExpression<float> *dExprMu, CudaExpression<float> *dExprSig);
template void fillFieldValue   <Material<float>>(Field<Material<float>> &dst, const Material<float> &val);
template void fillField        <float >         (Field<float>  &dst, CudaExpression<float > *dExpr);
template void fillField        <float3>         (Field<float3> &dst, CudaExpression<float3> *dExpr);
template void fillFieldChannel <float2>         (Field<float2> &dst, CudaExpression<float>  *dExpr, int channel);



