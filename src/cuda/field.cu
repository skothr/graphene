#include "hip/hip_runtime.h"
#include "field.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "physics.h"
#include "raytrace.cuh"
#include "vector-operators.h"
#include "cuda-tools.cuh"
#include "mathParser.hpp"

#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16
#define BLOCKDIM_Z 1


//// FILL FIELD ////

// fills field with constant value
template<typename T> __global__ void fieldFillValue_k(Field<T> dst, T val)
{
  long long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long long iy = blockIdx.y*blockDim.y + threadIdx.y;
  long long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix >= 0 && iy >= 0 && ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long long i = dst.idx(ix, iy, iz);
      dst[i] = val;
    }
}
// fills field via given math expression
template<typename T> __global__ void fillField_k(Field<T> dst, CudaExpression<T> *expr);
template<typename T> __global__ void fillFieldChannel_k(Field<T> dst, CudaExpression<typename Dims<T>::BASE> *expr, int channel=-1);
template<> __global__ void fillField_k<float>(Field<float> dst, CudaExpression<float> *expr)
{
  unsigned long long ix = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned long long iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned long long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long long i = dst.idx(ix, iy, iz);

      // allowed expression variables
      float3 s = float3{(float)dst.size.x, (float)dst.size.y, (float)dst.size.z}; // (s --> size)
      float3 p = float3{(float)ix, (float)iy, (float)iz};                         // (p --> position)
      float3 c = p - s/2.0;              // offset from field center
      float3 n = normalize(c);           // unit vector from field center
      float  r = length(c);              // (r --> radius) distance from field center
      float  t = (float)atan2(n.y, n.x); // (t --> theta)  angle measured from field center

      // run expression
      const int nVars = 8;
      float vars[nVars]; // {"px", "py", "pz", "sx", "sy", "sz", "r", "t"}
      vars[0] = p.x; vars[1] = p.y; vars[2] = p.z; // "px" / "py" / "pz"
      vars[3] = s.x; vars[4] = s.y; vars[5] = s.z; // "sx" / "sy" / "sz"
      vars[6] = r;   vars[7] = t;                  // "r" / "t"
      dst[i] = expr->calculate(vars);
    }
}
template<> __global__ void fillField_k<float3>(Field<float3> dst, CudaExpression<float3> *expr)
{
  long long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long long iy = blockIdx.y*blockDim.y + threadIdx.y;
  long long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long long i = dst.idx(ix, iy,iz);

      // allowed expression variables
      float3 s    = makeV<float3>(dst.size); //float3{(float)dst.size.x, (float)dst.size.y, 1.0}; // (s --> size)
      float3 p    = float3{(float)ix, (float)iy, (float)iz};  // (p --> position)
      float3 c    = p - s/2.0;       // offset from field center
      float3 n    = normalize(c);    // unit vector from field center
      float  r    = length(c);       // (r --> radius) distance from field center
      float  t    = atan2(n.y, n.x); // (t --> theta)  angle measured from field center (cylindrical)
      
      const int nVars = 5;
      float3 vars[nVars]; // {"p", "s", "r", "n", "t"}
      vars[0] = p;        // "p" -- position from origin (field index 0)
      vars[1] = s;        // "s" -- size
      vars[2] = c;        // "r" -- radius (position from center)
      vars[3] = n;        // "n" -- normalized radius
      //vars[4] = float3{t, t, t}; // "t" -- theta from center
      vars[4] = float3{atan2(n.x, n.y),
                       atan2(n.y, n.z),
                       atan2(n.z, n.x)}; // t alternative?
      // calculate value
      dst[i] = expr->calculate(vars);
    }
}
// only set one component/channel of each cell (used for setting +/- charge in Q.x/y
template<> __global__ void fillFieldChannel_k<float2>(Field<float2> dst, CudaExpression<float> *expr, int channel)
{
  unsigned long long ix = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned long long iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned long long iz = blockIdx.z*blockDim.z + threadIdx.z;
  if(ix < dst.size.x && iy < dst.size.y && iz < dst.size.z)
    {
      unsigned long long i = dst.idx(ix, iy, iz);
      // allowed expression variables
      float3 s    = float3{(float)dst.size.x, (float)dst.size.y, (float)dst.size.z}; // (s --> size)
      float3 p    = float3{(float)ix, (float)iy, (float)iz};           // (p --> position)
      float3 c    = p - s/2.0;                                         // offset from field center
      float3 n    = normalize(c);                                      // unit vector from field center
      float  r    = length(c);                                         // (r --> radius) distance from field center
      float  t    = atan2(n.y, n.x);                                   // (t --> theta)  angle measured from field center
      
      // run expression
      const int nVars = 8;
      float vars[nVars]; // {"px", "py", "pz", "sx", "sy", "sz", "r", "t"}
      vars[0] = p.x; vars[1] = p.y; vars[2] = p.z; // "px" / "py" / "pz"
      vars[3] = s.x; vars[4] = s.y; vars[5] = s.z; // "sx" / "sy" / "sz"
      vars[6] = r;   vars[7] = t;                  // "r" / "t"
      float val = expr->calculate(vars);

      // write to channel
      if     (channel == 0) { dst[i].x = val; }
      else if(channel == 1) { dst[i].y = val; }
      else                  { dst[i] = float2{val, val}; }
    }
}

// wrapppers
template<typename T>
void fieldFillValue(Field<T> &dst, const T &val)
{
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      fieldFillValue_k<<<grid, threads>>>(dst, val);
      hipDeviceSynchronize(); getLastCudaError("====> ERROR: fieldFillValue_k failed!");
    }
  else { std::cout << "Skipped Field<float> fill --> " << dst.size << " --> " << val << "\n"; }
}
template<typename T>
void fieldFill(Field<T> &dst, CudaExpression<T> *dExpr)
{
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      if(dExpr) { fillField_k<<<grid, threads>>>(dst, dExpr); }
      hipDeviceSynchronize(); getLastCudaError("====> ERROR: fillField_k failed!");
    }
  else { std::cout << "Skipped Field<float> fill --> " << dst.size << " \n"; }
}
template<typename T>
void fieldFillChannel(Field<T> &dst, CudaExpression<typename Dims<T>::BASE> *dExpr, int channel)
{
  if(dst.size.x > 0 && dst.size.y > 0)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(dst.size.x/(float)BLOCKDIM_X),
                (int)ceil(dst.size.y/(float)BLOCKDIM_Y),
                (int)ceil(dst.size.z/(float)BLOCKDIM_Z));
      if(dExpr) { fillFieldChannel_k<<<grid, threads>>>(dst, dExpr, channel); }
      hipDeviceSynchronize(); getLastCudaError("====> ERROR: fillFieldChannel_k failed!");
    }
  else { std::cout << "Skipped Field<float> fill --> " << dst.size << " \n"; }
}

// template instantiation
template void fieldFillValue  <Material<float>>(Field<Material<float>> &dst, const Material<float> &val);
template void fieldFill       <float>          (Field<float>  &dst, CudaExpression<float > *dExpr);
template void fieldFill       <float3>         (Field<float3> &dst, CudaExpression<float3> *dExpr);
template void fieldFillChannel<float2>         (Field<float2> &dst, CudaExpression<float>  *dExpr, int channel);








//// PHYSICS UPDATES ////

template<typename T>
__global__ void updateCharge_k(EMField<T> src, EMField<T> dst, FieldParams<T> cp)
{
  using VT2 = typename DimType<T, 2>::VECTOR_T;
  using VT3 = typename DimType<T, 3>::VECTOR_T;
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y;
  int iz = blockIdx.z*blockDim.z + threadIdx.z;

  if(ix < src.size.x && iy < src.size.y && iz < dst.size.z)
    {
      int3 p0 = int3{ix, iy, iz};
      int  i0 = src.idx(p0.x, p0.y, p0.z);

      VT3 E0  = src.E[i0];
      VT2 Q0  = VT2{src.QP[i0], src.QN[i0]};
      VT3 QVp0 = VT3{(ix < src.size.x-1 ? src.QVx[src.QVx.idx(ix, iy, iz)] : (T)0),
                     (iy < src.size.y-1 ? src.QVy[src.QVy.idx(ix, iy, iz)] : (T)0),
                     (iz < src.size.z-1 ? src.QVz[src.QVz.idx(ix, iy, iz)] : (T)0)};
      VT3 QVn0 = VT3{(ix > 0 ? src.QVx[src.QVx.idx(ix-1, iy, iz)] : (T)0),
                     (iy > 0 ? src.QVy[src.QVy.idx(ix, iy-1, iz)] : (T)0),
                     (iz > 0 ? src.QVz[src.QVz.idx(ix, iy, iz-1)] : (T)0)};
      Material<T> M0 = src.mat[i0]; if(M0.vacuum()) { M0 = cp.u.vacuum(); }
      
      //T   q0  = Q0.x - Q0.y; // total charge in cell (TODO: combine?)
      
      // // update velocities based on charge gradient
      VT2 newQ = Q0;
      // if(iPX < src.size.x) { T QV1 = src.Q[iPX]; newQ -= (q0-(Q1.x-Q1.y)); } //pv.x += (q0-(Q1.x-Q1.y)*(Q1.x+Q1.y))*cp.u.dt;
      // if(iNX >= 0)         { T QV1 = src.Q[iNX];  newQ -= (q0-(Q1.x-Q1.y)); } //nv.x += (q0-(Q1.x-Q1.y)*(Q1.x+Q1.y))*cp.u.dt;
      // if(iPY < src.size.y) { T QV1 = src.Q[iPY];  newQ -= (q0-(Q1.x-Q1.y)); } //pv.y += (q0-(Q1.x-Q1.y)*(Q1.x+Q1.y))*cp.u.dt;
      // if(iNY >= 0)         { T QV1 = src.Q[iNY];  newQ -= (q0-(Q1.x-Q1.y)); } //nv.y += (q0-(Q1.x-Q1.y)*(Q1.x+Q1.y))*cp.u.dt;
      // if(iPZ < src.size.z) { T QV1 = src.Q[iPZ];  newQ -= (q0-(Q1.x-Q1.y)); } //pv.z += (q0-(Q1.x-Q1.y)*(Q1.x+Q1.y))*cp.u.dt;
      // if(iNZ >= 0)         { T QV1 = src.Q[iNZ];  newQ -= (q0-(Q1.x-Q1.y)); } //nv.z += (q0-(Q1.x-Q1.y)*(Q1.x+Q1.y))*cp.u.dt;

      // TEST -- treat Q as fully derived from divergence of E (TODO: conserve?)
      int iPX = dst.E.idx(ix+(ix<src.size.x-1?1:0), iy, iz);
      int iNX = dst.E.idx(ix-(ix>0?1:0), iy, iz);
      int iPY = dst.E.idx(ix, iy+(iy<src.size.y-1?1:0), iz);
      int iNY = dst.E.idx(ix, iy-(iy>0?1:0), iz);
      int iPZ = dst.E.idx(ix, iy, iz+(iz<src.size.z-1?1:0));
      int iNZ = dst.E.idx(ix, iy, iz-(iz>0?1:0));
      VT3 gradE = VT3{0.0,0.0,0.0};
      if(ix < src.size.x-1) { gradE.x = src.E[iPX].x - E0.x;   }
      if(iy < src.size.y-1) { gradE.y = src.E[iPY].y - E0.y;   }
      if(iz < src.size.z-1) { gradE.z = src.E[iPZ].z - E0.z;   }

      newQ.x = sum(gradE) * cp.u.e0; // (~Gauss's law)
      //newQ.y = 0;
      
      dst.QP[i0]  = newQ.x;
      dst.QN[i0]  = newQ.y;
      if(ix < src.size.x-1) { dst.QVx[i0]  = QVp0.x; }
      if(iy < src.size.y-1) { dst.QVy[i0]  = QVp0.y; }
      if(iz < src.size.z-1) { dst.QVz[i0]  = QVp0.z; }
      dst.E[i0]   = E0;
      dst.B[i0]   = src.B[i0];
      dst.mat[i0] = src.mat[i0];
    }
}



//// SIMULATION -- MAXWELL'S EQUATIONS ////

// electric field E
template<typename T>
__global__ void updateElectric_k(EMField<T> src, EMField<T> dst, FieldParams<T> cp)
{
  using VT2 = typename DimType<T, 2>::VECTOR_T;
  using VT3 = typename DimType<T, 3>::VECTOR_T;
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y;
  int iz = blockIdx.z*blockDim.z + threadIdx.z;

  if(ix < src.size.x && iy < src.size.y && iz < dst.size.z)
    {
      int3 ip0 = int3{ix, iy, iz};
      int  i0  = src.idx(ix, iy, iz);
      VT3  p0  = makeV<VT3>(ip0)+VT3{0.5f, 0.5f, 0.5f};

      // check for boundary (TODO: not working? -- reflects)
      if(!cp.reflect) 
        {
          const int bs = 2;
          int xOffset = (src.size.x != 2*bs ? ((ip0.x < bs ? 1 : 0) + (ip0.x+bs >= src.size.x ? -1 : 0)) : 0);
          int yOffset = (src.size.y != 2*bs ? ((ip0.y < bs ? 1 : 0) + (ip0.y+bs >= src.size.y ? -1 : 0)) : 0);
          int zOffset = (src.size.z != 2*bs ? ((ip0.z < bs ? 1 : 0) + (ip0.z+bs >= src.size.z ? -1 : 0)) : 0);
          if(xOffset != 0 || yOffset != 0 || zOffset != 0)
            {
              int i = src.idx(max(0, min(src.size.x-1, ip0.x + xOffset)),
                              max(0, min(src.size.y-1, ip0.y + yOffset)),
                              max(0, min(src.size.z-1, ip0.z + zOffset)));
              dst.E[i0]   = src.E[i];  // use updated index for E
              dst.B[i0]   = src.B[i0]; // just copy everything else
              dst.QP[i0]  = src.QP[i0];
              dst.QN[i0]  = src.QN[i0];
              dst.QVx[dst.QVx.idx(ix,iy,iz)] = src.QVx[dst.QVx.idx(ix,iy,iz)];
              dst.QVy[dst.QVy.idx(ix,iy,iz)] = src.QVy[dst.QVy.idx(ix,iy,iz)];
              dst.QVz[dst.QVz.idx(ix,iy,iz)] = src.QVz[dst.QVz.idx(ix,iy,iz)];
              dst.mat[i0] = src.mat[i0];
              return;
            }
        }
      
      VT3 E0 = src.E[i0];
      VT3 B0 = src.B[i0];
      
      VT2 Q0  = VT2{src.QP[i0], src.QN[i0]};
      VT3 QVp0 = VT3{(ix < src.size.x-1 ? src.QVx[src.QVx.idx(ix, iy, iz)] : (T)0),
                     (iy < src.size.y-1 ? src.QVy[src.QVy.idx(ix, iy, iz)] : (T)0),
                     (iz < src.size.z-1 ? src.QVz[src.QVz.idx(ix, iy, iz)] : (T)0)};
      VT3 QVn0 = VT3{(ix > 0 ? src.QVx[src.QVx.idx(ix-1, iy, iz)] : (T)0),
                     (iy > 0 ? src.QVy[src.QVy.idx(ix, iy-1, iz)] : (T)0),
                     (iz > 0 ? src.QVz[src.QVz.idx(ix, iy, iz-1)] : (T)0)};
      Material<T> M0 = src.mat[i0];
      if(M0.vacuum()) { M0 = cp.u.vacuum(); } // check if vacuum
      typename Material<T>::Factors f = M0.getFactors(cp.u.dt, cp.u.dL);
      
      int3 ip1  = int3{min(src.size.x-1, max(0, ip0.x-1)), min(src.size.y-1, max(0, ip0.y-1)), min(src.size.z-1, max(0, ip0.z-1))};
      VT3  Bxn  = src.B[src.B.idx(ip1.x, ip0.y, ip0.z)]; // -1 in x direction
      VT3  Byn  = src.B[src.B.idx(ip0.x, ip1.y, ip0.z)]; // -1 in y direction
      VT3  Bzn  = src.B[src.B.idx(ip0.x, ip0.y, ip1.z)]; // -1 in z direction
      VT3  dEdt = VT3{  (B0.z-Byn.z) - (B0.y-Bzn.y),   // dBz/dY - dBy/dZ
                        (B0.x-Bzn.x) - (B0.z-Bxn.z),   // dBx/dZ - dBz/dX
                        (B0.y-Bxn.y) - (B0.x-Byn.x) }; // dBy/dX - dBx/dY

      // apply effect of electric current (TODO: improve)
      VT3 dS = VT3{cp.u.dL*cp.u.dL, cp.u.dL*cp.u.dL, cp.u.dL*cp.u.dL};
      VT3 J = (QVp0-QVn0)*(Q0.x - Q0.y) / cp.u.dt / dS;
      dEdt -= J / M0.permittivity;
      
      VT3 newE = f.alphaE*E0 + f.betaE*dEdt;
      // TODO: solve for divergence?
      if(isnan(newE.x) || isinf(newE.x) || isnan(newE.y) || isinf(newE.y) || isnan(newE.z) || isinf(newE.z) ||
         abs(newE.x) > 1e24 ||abs(newE.y) > 1e24 ||abs(newE.z) > 1e24) { newE = VT3{0.0,0.0,0.0}; }
      
      // // lorentz (E)
      VT3 newQV = QVp0 + (Q0.x-Q0.y)*newE*cp.u.dt;
      if(isnan(newQV.x) || isinf(newQV.x) || isnan(newQV.y) || isinf(newQV.y) || isnan(newQV.z) || isinf(newQV.z)) { newQV = VT3{0.0,0.0,0.0}; }
      
      dst.E[i0]   = newE; // updated values
      if(ix > src.size.x) { dst.QVx[dst.QVx.idx(ix,iy,iz)] = newQV.x; }
      if(iy > src.size.y) { dst.QVy[dst.QVy.idx(ix,iy,iz)] = newQV.y; }
      if(iz > src.size.z) { dst.QVz[dst.QVz.idx(ix,iy,iz)] = newQV.z; }
    
      dst.QP[i0]  = Q0.x; dst.QN[i0] = Q0.y; // copied values (unchanged)
      
      dst.B[i0]   = B0;
      dst.mat[i0] = M0;
    }
}

// magnetic field B
template<typename T>
__global__ void updateMagnetic_k(EMField<T> src, EMField<T> dst, FieldParams<T> cp)
{
  using VT2 = typename DimType<T, 2>::VECTOR_T;
  using VT3 = typename DimType<T, 3>::VECTOR_T;
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y;
  int iz = blockIdx.z*blockDim.z + threadIdx.z;

  if(ix < src.size.x && iy < src.size.y && iz < dst.size.z)
    {
      int3  ip0 = int3{ix, iy, iz};
      int    i0 = src.idx(ix, iy, iz);

      // check for boundary (TODO: remove/fi)x
      if(!cp.reflect)
        {
          const int bs = 2;
          int xOffset = (src.size.x != 2*bs ? ((ip0.x < bs ? 1 : 0) + (ip0.x+bs >= src.size.x ? -1 : 0)) : 0);
          int yOffset = (src.size.y != 2*bs ? ((ip0.y < bs ? 1 : 0) + (ip0.y+bs >= src.size.y ? -1 : 0)) : 0);
          int zOffset = (src.size.z != 2*bs ? ((ip0.z < bs ? 1 : 0) + (ip0.z+bs >= src.size.z ? -1 : 0)) : 0);
          if(xOffset != 0 || yOffset != 0 || zOffset != 0)
            {
              int i = src.idx(max(0, min(src.size.x-1, ip0.x + xOffset)),
                              max(0, min(src.size.y-1, ip0.y + yOffset)),
                              max(0, min(src.size.z-1, ip0.z + zOffset)));
              dst.B[i0]   = src.B[i];  // use updated index for B
              dst.E[i0]   = src.E[i0]; // just copy everything else
              dst.QP[i0]  = src.QP[i0];
              dst.QN[i0]  = src.QN[i0];
              dst.QVx[dst.QVx.idx(ix,iy,iz)] = src.QVx[dst.QVx.idx(ix,iy,iz)];
              dst.QVy[dst.QVy.idx(ix,iy,iz)] = src.QVy[dst.QVy.idx(ix,iy,iz)];
              dst.QVz[dst.QVz.idx(ix,iy,iz)] = src.QVz[dst.QVz.idx(ix,iy,iz)];
              dst.mat[i0] = src.mat[i0];
              return;
            }
        }
      
      VT3 B0         = src.B[i0];
      VT3 E0         = src.E[i0];
      VT2 Q0  = VT2{src.QP[i0], src.QN[i0]};
      VT3 QVp0 = VT3{(ix < src.size.x-1 ? src.QVx[src.QVx.idx(ix, iy, iz)] : (T)0),
                     (iy < src.size.y-1 ? src.QVy[src.QVy.idx(ix, iy, iz)] : (T)0),
                     (iz < src.size.z-1 ? src.QVz[src.QVz.idx(ix, iy, iz)] : (T)0)};
      VT3 QVn0 = VT3{(ix > 0 ? src.QVx[src.QVx.idx(ix-1, iy, iz)] : (T)0),
                     (iy > 0 ? src.QVy[src.QVy.idx(ix, iy-1, iz)] : (T)0),
                     (iz > 0 ? src.QVz[src.QVz.idx(ix, iy, iz-1)] : (T)0)};
      Material<T> M0 = src.mat[i0];
      if(M0.vacuum()) { M0 = cp.u.vacuum(); } // check if vacuum
      typename Material<T>::Factors f = M0.getFactors(cp.u.dt, cp.u.dL);

      int3 ip1  = int3{max(0, min(src.size.x-1, ip0.x+1)), max(0, min(src.size.y-1, ip0.y+1)), max(0, min(src.size.z-1, ip0.z+1)) };
      VT3  Exp  = src.E[src.E.idx(ip1.x, ip0.y, ip0.z)]; // +1 in x direction
      VT3  Eyp  = src.E[src.E.idx(ip0.x, ip1.y, ip0.z)]; // +1 in y direction
      VT3  Ezp  = src.E[src.E.idx(ip0.x, ip0.y, ip1.z)]; // +1 in z direction
      VT3  dBdt = VT3{  (Eyp.z-E0.z) - (Ezp.y-E0.y),   // dEz/dY - dEy/dZ
                        (Ezp.x-E0.x) - (Exp.z-E0.z),   // dEx/dZ - dEz/dX
                        (Exp.y-E0.y) - (Eyp.x-E0.x) }; // dEy/dX - dEx/dY
      VT3 newB = f.alphaB*B0 - f.betaB*dBdt;
      if(isnan(newB.x) || isinf(newB.x) || isnan(newB.y) || isinf(newB.y) || isnan(newB.z) || isinf(newB.z) ||
         abs(newB.x) > 1e24 ||abs(newB.y) > 1e24 ||abs(newB.z) > 1e24) { newB = VT3{0.0,0.0,0.0}; }
      
      // // lorentz (v x B)
      VT3 newQV = QVp0 + (Q0.x-Q0.y)*cross(QVp0, newB)*cp.u.dt;
      if(isnan(newQV.x) || isinf(newQV.x) || isnan(newQV.y) || isinf(newQV.y) || isnan(newQV.z) || isinf(newQV.z)) { newQV = VT3{0.0,0.0,0.0}; }
        
      dst.B[i0]   = newB; // updated values
      if(ix > src.size.x) { dst.QVx[dst.QVx.idx(ix,iy,iz)] = newQV.x; }
      if(iy > src.size.y) { dst.QVy[dst.QVy.idx(ix,iy,iz)] = newQV.y; }
      if(iz > src.size.z) { dst.QVz[dst.QVz.idx(ix,iy,iz)] = newQV.z; }
    
      dst.QP[i0]  = Q0.x; // copied values (unchanged)
      dst.QN[i0]  = Q0.y;
      dst.E[i0]   = E0;
      dst.mat[i0] = M0;
    }
}

// wrappers
template<typename T> void updateCharge(EMField<T> &src, EMField<T> &dst, FieldParams<T> &cp)
{
  if(src.size.x > 0 && src.size.y > 0 && dst.size == src.size)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(src.size.x/(float)BLOCKDIM_X),
                (int)ceil(src.size.y/(float)BLOCKDIM_Y),
                (int)ceil(src.size.z/(float)BLOCKDIM_Z));
      updateCharge_k<<<grid, threads>>>(src, dst, cp);
      hipDeviceSynchronize(); getLastCudaError("====> ERROR: updateCharge_k failed!");
    }
  else { std::cout << "==> WARNING: Skipped updateCharge (" << src.size << " / " << dst.size << ")\n"; }
}
template<typename T> void updateElectric(EMField<T> &src, EMField<T> &dst, FieldParams<T> &cp)
{
  if(src.size.x > 0 && src.size.y > 0 && dst.size == src.size)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(src.size.x/(float)BLOCKDIM_X),
                (int)ceil(src.size.y/(float)BLOCKDIM_Y),
                (int)ceil(src.size.z/(float)BLOCKDIM_Z));
      updateElectric_k<<<grid, threads>>>(src, dst, cp);
      hipDeviceSynchronize(); getLastCudaError("====> ERROR: updateElectric_k failed!");
    }
  else { std::cout << "==> WARNING: Skipped updateElectric (" << src.size << " / " << dst.size << ")\n"; }
}
template<typename T> void updateMagnetic(EMField<T> &src, EMField<T> &dst, FieldParams<T> &cp)
{
  if(src.size.x > 0 && src.size.y > 0 && dst.size == src.size)
    {
      dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
      dim3 grid((int)ceil(src.size.x/(float)BLOCKDIM_X),
                (int)ceil(src.size.y/(float)BLOCKDIM_Y),
                (int)ceil(src.size.z/(float)BLOCKDIM_Z));
      updateMagnetic_k<<<grid, threads>>>(src, dst, cp);
      hipDeviceSynchronize(); getLastCudaError("====> ERROR: updateMagnetic2D_k failed!");
    }
  else { std::cout << "==> WARNING: Skipped updateMagnetic2D (src: " << src.size << " / dst: " << dst.size << ")\n"; }
}

// template instantiation
template void updateCharge  <float> (EMField<float> &src, EMField<float>  &dst, FieldParams<float> &cp);
template void updateElectric<float> (EMField<float> &src, EMField<float>  &dst, FieldParams<float> &cp);
template void updateMagnetic<float> (EMField<float> &src, EMField<float>  &dst, FieldParams<float> &cp);
// template void updateCharge  <double>(EMField<double> &src, EMField<double> &dst, FieldParams<double> &cp);
// template void updateElectric<double>(EMField<double> &src, EMField<double> &dst, FieldParams<double> &cp);






